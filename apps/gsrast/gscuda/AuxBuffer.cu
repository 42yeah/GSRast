#include "AuxBuffer.cuh"
#include <hipcub/hipcub.hpp>


namespace gscuda
{
    /**
     * We need to align because CUDA **WILL** complain 
     * if the memory is not properly aligned to - I think at least 16 bytes?
     * chunk is both the input and output variable.
     * out is the output pointer obtained from said chunk.
     */
    template<typename T>
    void obtain(char *&chunk, T *&out, size_t size, size_t align)
    {
        assert(align >= 16 && "CUDA requires an alignment of at least 16 bytes");
        size_t offset = reinterpret_cast<size_t>(chunk);
        size_t aligned = align * ((offset + align - 1) / align);
        out = reinterpret_cast<T *>(aligned);
        chunk = reinterpret_cast<char *>(aligned + size);
    }

    namespace pc
    {
        GeometryState GeometryState::fromChunk(char *&chunk, int numGaussians)
        {
            GeometryState state;
            return state;
        }

        ImageState ImageState::fromChunk(char *&chunk, int size)
        {
            ImageState state;
            obtain(chunk, state.depth, sizeof(float) * size, 128);
            obtain(chunk, state.outColor, sizeof(float) * size * 3, 128);
            obtain(chunk, state.defaultDepth, sizeof(float), 128);

            return state;
        }
    }

    namespace gs
    {
        GeometryState GeometryState::fromChunk(char *&chunk, int numGaussians)
        {
            GeometryState state;

            obtain(chunk, state.tilesTouched, sizeof(uint32_t) * numGaussians, 128);
            hipcub::DeviceScan::InclusiveSum(nullptr, state.scanSize, state.tilesTouched, state.tilesTouched, numGaussians);
            state.numRendered = 0;
            obtain(chunk, state.scanningSpace, state.scanSize, 128);

            obtain(chunk, state.depths, sizeof(float) * numGaussians, 128);
            obtain(chunk, state.clamped, sizeof(bool) * numGaussians * 3, 128);
            obtain(chunk, state.internalRadii, sizeof(float) * numGaussians, 128);
            obtain(chunk, state.means2D, sizeof(glm::vec2) * numGaussians, 128);
            obtain(chunk, state.cov3D, 6 * sizeof(float) * numGaussians, 128); // Upper-right corner of the matrix (because it's symmetric)
            obtain(chunk, state.conicOpacity, sizeof(glm::vec4) * numGaussians, 128);
            obtain(chunk, state.rgb, sizeof(glm::vec3) * numGaussians, 128);
            obtain(chunk, state.pointOffsets, sizeof(uint32_t) * numGaussians, 128);

	    obtain(chunk, state.ellipsoids, sizeof(MathematicalEllipsoid) * numGaussians, 128);
	    obtain(chunk, state.ellipses, sizeof(MathematicalEllipse) * numGaussians, 128);

            return state;
        }

        ImageState ImageState::fromChunk(char *&chunk, int size)
        {
            ImageState state;
            obtain(chunk, state.ranges, sizeof(glm::uvec2) * size, 128);
            obtain(chunk, state.nContrib, sizeof(uint32_t) * size, 128);
            obtain(chunk, state.accumAlpha, sizeof(float) * size, 128);

            return state;
        }

        BinningState BinningState::fromChunk(char *&chunk, int size)
        {
            BinningState state;
            obtain(chunk, state.pointListKeysUnsorted, sizeof(uint64_t) * size, 128);
            obtain(chunk, state.pointListKeys, sizeof(uint64_t) * size, 128);
            obtain(chunk, state.pointListUnsorted, sizeof(uint32_t) * size, 128);
            obtain(chunk, state.pointList, sizeof(uint32_t) * size, 128);
            // Estimate sorting size
            hipcub::DeviceRadixSort::SortPairs(nullptr, state.sortingSize,
                                            state.pointListKeysUnsorted, state.pointListKeys,
                                            state.pointListUnsorted, state.pointList, size);
            obtain(chunk, state.listSortingSpace, state.sortingSize, 128);

            return state;
        }
    }
}

